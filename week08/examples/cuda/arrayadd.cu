
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(const float * input1, const float * input2, float * output, size_t len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < len)
    {
        printf("(%d, %d) working for element %d\n", blockDim.x, threadIdx.x, i);
        output[i] = input1[i] + input2[i];
    }
    else
    {
        printf("(%d, %d) skipped element %d\n", blockDim.x, threadIdx.x, i);
    }

}

int main()
{
    hipError_t ecode = hipSuccess;
    const size_t len = 8;
    float vec1[len]{0.f, 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f};
    float vec2[len]{1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f, 8.f};
    float vec3[len]{};

    float * cudaVec1 = NULL;
    float * cudaVec2 = NULL;
    float * cudaVec3 = NULL;


    if (hipMalloc (&cudaVec1, sizeof(float) * len) != hipSuccess
        || hipMalloc (&cudaVec2, sizeof(float) * len) != hipSuccess
        || hipMalloc (&cudaVec3, sizeof(float) * len) != hipSuccess)
    {
        fprintf(stderr, "Allocate device memory failed.\n");
        return -1;
    }
    hipMemcpy(cudaVec1, vec1, sizeof(float)*len, hipMemcpyHostToDevice);
    hipMemcpy(cudaVec2, vec2, sizeof(float)*len, hipMemcpyHostToDevice);
    add<<<2,2>>>(cudaVec1, cudaVec2, cudaVec3, len);
    if ((ecode = hipGetLastError()) != hipSuccess) // check 
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(ecode));
        return false;
    }
    hipMemcpy(vec3, cudaVec3, sizeof(float)*len, hipMemcpyDeviceToHost);  
    hipDeviceSynchronize();

    for(size_t i = 0; i < len; i++)
        printf("vec3[%zu] = %.1f\n", i, vec3[i]);

    return 0;
}