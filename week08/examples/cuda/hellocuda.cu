
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel()
{
    printf("Hello, CUDA. I'm Thread %d in Block %d. blockDim=%d.\n",
            threadIdx.x, blockIdx.x, blockDim.x);
}

int main()
{
    hipError_t ecode = hipSuccess;
    mykernel<<<2,2>>>();
    if ((ecode = hipGetLastError()) != hipSuccess) // check 
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(ecode));
        return false;
    }
    hipDeviceSynchronize();
    
    return 0;
}