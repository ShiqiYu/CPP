
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h> 

#define TIME_START gettimeofday(&t_start, NULL);
#define TIME_END(name)    gettimeofday(&t_end, NULL); \
                    elapsedTime = (t_end.tv_sec - t_start.tv_sec) * 1000.0;   \
                    elapsedTime += (t_end.tv_usec - t_start.tv_usec) / 1000.0;  \
                    printf(#name " Time = %f ms.\n", elapsedTime);

typedef struct 
{
    size_t rows;
    size_t cols;
    float * data; // CPU memory
    float * data_device; //GPU mememory
} Matrix;

Matrix * createMatrix(size_t r, size_t c)
{
    size_t len = r * c;
    if(len == 0)
    {
        fprintf(stderr, "Invalid size. The input should be > 0.\n");
        return NULL;
    }
    Matrix * p  = (Matrix *) malloc(sizeof(Matrix));
    if (p == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    p->rows = r;
    p->cols = c;
    p->data = (float*)malloc(sizeof(float)*len);
    if(p->data == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    if (hipMalloc (&p->data_device, sizeof(float) * len) != hipSuccess)
    {
        fprintf(stderr, "Allocate device memory failed.\n");
        goto ERR_TAG;
    }
    return p;
  ERR_TAG:
    if(p && p->data) free(p->data);
    if(p) free(p);
    return NULL;
}

void freeMatrix(Matrix ** pp)
{
    if(pp == NULL) return;
    Matrix * p = *pp;
    if(p != NULL)
    {
        if(p->data) free(p->data);
        if(p->data_device) hipFree(p->data_device);
    }
    *pp = NULL;
}
// a simple function to set all elements to the same value
bool setMatrix(Matrix * pMat, float val)
{
    if(pMat == NULL)
    {
        fprintf(stderr, "NULL pointer.\n");
        return false;
    }
    size_t len = pMat->rows * pMat->cols;
    for(size_t i = 0; i < len; i++)
        pMat->data[i] = val;

    return true;
}


bool addCPU(const Matrix * pMat1, const Matrix * pMat2, Matrix * pMat3)
{
    if( pMat1 == NULL 
        || pMat2 == NULL
        || pMat3 == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMat1->rows != pMat2->rows || pMat1->cols != pMat2->cols ||
        pMat2->rows != pMat3->rows || pMat2->cols != pMat3->cols)
    {
        fprintf(stderr, "The 3 matrics are not in the same size.\n");
        return false;
    }
    size_t len = pMat1->rows * pMat1->cols;
    for (int i = 0; i < len; i++)
        pMat3->data[i] = pMat1->data[i] + pMat2->data[i];
    return true;
}

__global__ void addKernel(const float * input1, const float * input2, float * output, size_t len)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < len)
	    output[i] = input1[i] + input2[i];
}

bool addGPU(const Matrix * pMat1, const Matrix * pMat2, Matrix * pMat3)
{
    if( pMat1 == NULL 
        || pMat2 == NULL
        || pMat3 == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMat1->rows != pMat2->rows || pMat1->cols != pMat2->cols ||
        pMat2->rows != pMat3->rows || pMat2->cols != pMat3->cols)
    {
        fprintf(stderr, "The 3 matrics are not in the same size.\n");
        return false;
    }

    hipError_t ecode = hipSuccess;
    size_t len = pMat1->rows * pMat1->cols;

    hipMemcpy(pMat1->data_device, pMat1->data, sizeof(float)*len, hipMemcpyHostToDevice);
    hipMemcpy(pMat2->data_device, pMat2->data, sizeof(float)*len, hipMemcpyHostToDevice);
    addKernel<<<(len+255)/256, 256>>>(pMat1->data_device, pMat2->data_device, pMat3->data_device, len);
    if ((ecode = hipGetLastError()) != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(ecode));
        return false;
    }
    hipMemcpy(pMat3->data, pMat3->data_device, sizeof(float)*len, hipMemcpyDeviceToHost);

    return true;
}

int main()
{

    struct timeval t_start, t_end;
    double elapsedTime = 0;

    int dev_count = 0;
    int dev_id = 0;
    hipGetDeviceCount(&dev_count);
    hipSetDevice(2);
    hipGetDevice(&dev_id);
    printf("You have %d cuda devices.\n", dev_count);
    printf("You are using device %d.\n", dev_id);

    Matrix * pMat1 = createMatrix(4096, 4096);
    Matrix * pMat2 = createMatrix(4096, 4096);
    Matrix * pMat3 = createMatrix(4096, 4096);

    setMatrix(pMat1, 1.1f);
    setMatrix(pMat2, 2.2f);

    TIME_START
    addCPU(pMat1, pMat2, pMat3);
    TIME_END(addCPU)
    printf("  Result = [%.1f, ..., %.1f]\n", pMat3->data[0], pMat3->data[pMat3->rows*pMat3->cols-1]);

    TIME_START
    addGPU(pMat1, pMat2, pMat3);
    TIME_END(addGPU)
    printf("  Result = [%.1f, ..., %.1f]\n", pMat3->data[0], pMat3->data[pMat3->rows*pMat3->cols-1]);

    freeMatrix(&pMat1);
    freeMatrix(&pMat2);
    freeMatrix(&pMat3);
    return 0;
}